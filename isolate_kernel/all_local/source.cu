#include "hip/hip_runtime.h"
#include "unused.cuh"

template <typename ValueType>
__global__ void dummy(int num, ValueType* val) {
    auto tid = get_index();
    val[threadIdx.x] += 1;
    
}

int main () {
    float* d_A;
    float data[64];
    for (int i = 0; i < 64; i++) {
        // 0~15: 0, 16~31: 1
        data[i] = i / 16;
    }
    hipMalloc(&d_A, 64 * sizeof(float));
    hipMemcpy(d_A, data, 64 * sizeof(float), hipMemcpyHostToDevice);
    dummy<<<1, 64>>>(64, d_A);
    hipDeviceSynchronize();
    hipMemcpy(data, d_A, 64 * sizeof(float), hipMemcpyDeviceToHost);
    return 0;
}